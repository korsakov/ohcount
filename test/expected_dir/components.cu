#include "hip/hip_runtime.h"
cuda	code	#include <unistd.h>
cuda	code	#include <error.h>
cuda	code	#include <stdio.h>
cuda	code	#include <stdlib.h>
cuda	code	#include <errno.h>
cuda	code	#include <assert.h>
cuda	blank	
cuda	code	#include "components.h"
cuda	code	#include "common.h"
cuda	blank	
cuda	code	#define THREADS 256
cuda	blank	
cuda	comment	/* Store 3 RGB float components */
cuda	code	__device__ void storeComponents(float *d_r, float *d_g, float *d_b, float r, float g, float b, int pos)
cuda	code	{
cuda	code	    d_r[pos] = (r/255.0f) - 0.5f;
cuda	code	    d_g[pos] = (g/255.0f) - 0.5f;
cuda	code	    d_b[pos] = (b/255.0f) - 0.5f;
cuda	code	}
cuda	blank	
cuda	comment	/* Store 3 RGB intege components */
cuda	code	__device__ void storeComponents(int *d_r, int *d_g, int *d_b, int r, int g, int b, int pos)
cuda	code	{
cuda	code	    d_r[pos] = r - 128;
cuda	code	    d_g[pos] = g - 128;
cuda	code	    d_b[pos] = b - 128;
cuda	code	}
cuda	blank	
cuda	comment	/* Store float component */
cuda	code	__device__ void storeComponent(float *d_c, float c, int pos)
cuda	code	{
cuda	code	    d_c[pos] = (c/255.0f) - 0.5f;
cuda	code	}
cuda	blank	
cuda	comment	/* Store integer component */
cuda	code	__device__ void storeComponent(int *d_c, int c, int pos)
cuda	code	{
cuda	code	    d_c[pos] = c - 128;
cuda	code	}
cuda	blank	
cuda	comment	/* Copy img src data into three separated component buffers */
cuda	code	template<typename T>
cuda	code	__global__ void c_CopySrcToComponents(T *d_r, T *d_g, T *d_b, 
cuda	code	                                  unsigned char * d_src, 
cuda	code	                                  int pixels)
cuda	code	{
cuda	code	    int x  = threadIdx.x;
cuda	code	    int gX = blockDim.x*blockIdx.x;
cuda	blank	
cuda	code	    __shared__ unsigned char sData[THREADS*3];
cuda	blank	
cuda	comment	/* Copy data to shared mem by 4bytes other checks are not necessary, since d_src buffer is aligned to sharedDataSize */
cuda	code	    if ( (x*4) < THREADS*3 ) {
cuda	code	        float *s = (float *)d_src;
cuda	code	        float *d = (float *)sData;
cuda	code	        d[x] = s[((gX*3)>>2) + x];
cuda	code	    }
cuda	code	    __syncthreads();
cuda	blank	
cuda	code	    T r, g, b;
cuda	blank	
cuda	code	    int offset = x*3;
cuda	code	    r = (T)(sData[offset]);
cuda	code	    g = (T)(sData[offset+1]);
cuda	code	    b = (T)(sData[offset+2]);
cuda	blank	
cuda	code	    int globalOutputPosition = gX + x;
cuda	code	    if (globalOutputPosition < pixels) {
cuda	code	        storeComponents(d_r, d_g, d_b, r, g, b, globalOutputPosition);
cuda	code	    }
cuda	code	}
cuda	blank	
cuda	comment	/* Copy img src data into three separated component buffers */
cuda	code	template<typename T>
cuda	code	__global__ void c_CopySrcToComponent(T *d_c, unsigned char * d_src, int pixels)
cuda	code	{
cuda	code	    int x  = threadIdx.x;
cuda	code	    int gX = blockDim.x*blockIdx.x;
cuda	blank	
cuda	code	    __shared__ unsigned char sData[THREADS];
cuda	blank	
cuda	comment	/* Copy data to shared mem by 4bytes other checks are not necessary, since d_src buffer is aligned to sharedDataSize */
cuda	code	    if ( (x*4) < THREADS) {
cuda	code	        float *s = (float *)d_src;
cuda	code	        float *d = (float *)sData;
cuda	code	        d[x] = s[(gX>>2) + x];
cuda	code	    }
cuda	code	    __syncthreads();
cuda	blank	
cuda	code	    T c;
cuda	blank	
cuda	code	    c = (T)(sData[x]);
cuda	blank	
cuda	code	    int globalOutputPosition = gX + x;
cuda	code	    if (globalOutputPosition < pixels) {
cuda	code	        storeComponent(d_c, c, globalOutputPosition);
cuda	code	    }
cuda	code	}
cuda	blank	
cuda	blank	
cuda	comment	/* Separate compoents of 8bit RGB source image */
cuda	code	template<typename T>
cuda	code	void rgbToComponents(T *d_r, T *d_g, T *d_b, unsigned char * src, int width, int height)
cuda	code	{
cuda	code	    unsigned char * d_src;
cuda	code	    int pixels      = width*height;
cuda	code	    int alignedSize =  DIVANDRND(width*height, THREADS) * THREADS * 3; //aligned to thread block size -- THREADS
cuda	blank	
cuda	comment	    /* Alloc d_src buffer */
cuda	code	    hipMalloc((void **)&d_src, alignedSize);
cuda	code	    cudaCheckAsyncError("Cuda malloc")
cuda	code	    hipMemset(d_src, 0, alignedSize);
cuda	blank	
cuda	comment	     /* Copy data to device */
cuda	code	    hipMemcpy(d_src, src, pixels*3, hipMemcpyHostToDevice);
cuda	code	    cudaCheckError("Copy data to device")
cuda	blank	
cuda	comment	/* Kernel */
cuda	code	    dim3 threads(THREADS);
cuda	code	    dim3 grid(alignedSize/(THREADS*3));
cuda	code	    assert(alignedSize%(THREADS*3) == 0);
cuda	code	    c_CopySrcToComponents<<<grid, threads>>>(d_r, d_g, d_b, d_src, pixels);
cuda	code	    cudaCheckAsyncError("CopySrcToComponents kernel")
cuda	blank	
cuda	comment	/* Free Memory */
cuda	code	    hipFree(d_src);
cuda	code	    cudaCheckAsyncError("Free memory")
cuda	code	}
cuda	code	template void rgbToComponents<float>(float *d_r, float *d_g, float *d_b, unsigned char * src, int width, int height);
cuda	code	template void rgbToComponents<int>(int *d_r, int *d_g, int *d_b, unsigned char * src, int width, int height);
cuda	blank	
cuda	blank	
cuda	comment	/* Copy a 8bit source image data into a color compoment of type T */
cuda	code	template<typename T>
cuda	code	void bwToComponent(T *d_c, unsigned char * src, int width, int height)
cuda	code	{
cuda	code	    unsigned char * d_src;
cuda	code	    int pixels      = width*height;
cuda	code	    int alignedSize =  DIVANDRND(pixels, THREADS) * THREADS; //aligned to thread block size -- THREADS
cuda	blank	
cuda	comment	/* Alloc d_src buffer */
cuda	code	    hipMalloc((void **)&d_src, alignedSize);
cuda	code	    cudaCheckAsyncError("Cuda malloc")
cuda	code	    hipMemset(d_src, 0, alignedSize);
cuda	blank	
cuda	comment	/* Copy data to device */
cuda	code	    hipMemcpy(d_src, src, pixels, hipMemcpyHostToDevice);
cuda	code	    cudaCheckError("Copy data to device")
cuda	blank	
cuda	comment	/* Kernel */
cuda	code	    dim3 threads(THREADS);
cuda	code	    dim3 grid(alignedSize/(THREADS));
cuda	code	    assert(alignedSize%(THREADS) == 0);
cuda	code	    c_CopySrcToComponent<<<grid, threads>>>(d_c, d_src, pixels);
cuda	code	    cudaCheckAsyncError("CopySrcToComponent kernel")
cuda	blank	
cuda	comment	/* Free Memory */
cuda	code	    hipFree(d_src);
cuda	code	    cudaCheckAsyncError("Free memory")
cuda	code	}
cuda	blank	
cuda	code	template void bwToComponent<float>(float *d_c, unsigned char *src, int width, int height);
cuda	code	template void bwToComponent<int>(int *d_c, unsigned char *src, int width, int height);
